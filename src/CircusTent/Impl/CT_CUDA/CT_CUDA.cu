/*
 * _CT_CUDA_CU_
 *
 * Copyright (C) 2017-2021 Tactical Computing Laboratories, LLC
 * All Rights Reserved
 * contact@tactcomplabs.com
 *
 * See LICENSE in the top level directory for licensing details
 */

#include "CT_CUDA.cuh"
#ifdef _CT_CUDA_CUH_

CT_CUDA::CT_CUDA(CTBaseImpl::CTBenchType B, CTBaseImpl::CTAtomType A) :
    CTBaseImpl("CUDA", B, A),
    Array(nullptr),
    Idx(nullptr),
    d_Array(nullptr),
    d_Idx(nullptr),
    memSize(0),
    pes(0),
    iters(0),
    elems(0),
    stride(0),
    deviceID(-1),
    deviceCount(0),
    blocksPerGrid(-1),
    threadsPerBlock(-1)
    {}

CT_CUDA::~CT_CUDA() {}

// helper functions
bool CT_CUDA::PrintCUDADeviceProperties(int deviceID, int deviceCount) {
    hipGetDeviceCount(&deviceCount);
    // std::cout << "====================================================================================" << std::endl;
    // std::cout << "                             CUDA Device Properties"          << std::endl;
    // std::cout << "====================================================================================" << std::endl;
    std::cout << "\nNumber of CUDA enabled devices detected: " << deviceCount << std::endl;
    if (getenv("CUDA_VISIBLE_DEVICES") == nullptr) {
        std::cout << "CUDA_VISIBLE_DEVICES environment variable not set, defaulting to hipSetDevice(1)\n" << std::endl;
        deviceID = hipSetDevice(1);
    }

    if (!deviceID && getenv("CUDA_VISIBLE_DEVICES") == nullptr) {
        std::cout << "No target devices detected!" << std::endl;
        return false;
    }
    // else {
    //         hipDeviceProp_t prop;
    //         hipGetDeviceProperties(&prop, deviceID);
    //         std::cout << "Target CUDA deviceID : " << deviceID << std::endl;
    //         std::cout << "Device Name: " << prop.name << std::endl;
    //         std::cout << "Total Global Memory: " << prop.totalGlobalMem << std::endl;
    //         std::cout << "Memory Clock Rate (MHz): " << prop.memoryClockRate/1024 << std::endl;
    //         std::cout << "Maximum Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
    //         std::cout << "Warp Size: " << prop.warpSize << std::endl;
    //     }
        // std::cout << "" << std::endl;
        return true;
}

bool CT_CUDA::ParseCUDAOpts(int argc, char **argv) {
    for (int i=1; i < argc; i++) {
        std::string s(argv[i]);

        if ( (s=="-bpg") || (s=="-blocks") || (s=="--blocks") ) {
            if ( i+1 > (argc-1) ) {
                std::cout << "Error: --blocks requires an argument" << std::endl;
                return false;
            }
            std::string P(argv[i+1]);
            blocksPerGrid = atoi(P.c_str());
            i++;
        }
        else if ((s=="-tpb") || (s=="-threads") || (s=="--threads")) {
            if ( i+1 > (argc-1) ) {
                std::cout << "Error: --threads requires an argument" << std::endl;
                return false;
            }
            std::string P(argv[i+1]);
            threadsPerBlock = atoi(P.c_str());
            i++;
        }
    }

    // sanity check the options
    if ( blocksPerGrid <= 0 ) {
        std::cout << "Error: --blocks must be greater than 0" << std::endl;
        return false;
    }
    if ( threadsPerBlock <= 0 ) {
        std::cout << "Error: --threads must be greater than 0" << std::endl;
        return false;
    }

    return true;
}

bool CT_CUDA::AllocateData(uint64_t m, uint64_t p, uint64_t i, uint64_t s) {
    // save the data
    memSize = m;
    pes = p;
    iters = i;
    stride = s;
    uint64_t idxMemSize = 2 * memSize;

    // check args
    if ( pes == 0 ) {
        std::cout << "CT_CUDA::AllocateData: `pes` cannot be 0" << std::endl;
        return false;
    }
    if ( iters == 0 ) {
        std::cout << "CT_CUDA::AllocateData `iters` cannot be 0" << std::endl;
        return false;
    }
    if ( stride == 0 ) {
        std::cout << "CT_CUDA::AllocateData `stride` cannot be 0" << std::endl;
        return false;
    }

    // calculate the number of elements
    elems = (memSize/8);
    uint64_t idxElems = (idxMemSize/8);

    // test to see whether we'll stride out of bounds
    uint64_t end = (pes * iters * stride) - stride;
    if ( end > elems ) {
        std::cout << "CT_CUDA::AllocateData : `Array` is not large enough for pes="
        << pes << "; iters=" << iters << "; stride=" << stride << std::endl;
        return false;
    }

    // Allocate arrays on the host  
    Array = (uint64_t *) malloc(memSize);
    if ( Array == nullptr ) {
        std::cout << "CT_CUDA::AllocateData : 'Array' could not be allocated" << std::endl;
        free(Array);
        return false;
    }

    Idx = (uint64_t *) malloc(idxMemSize);
    if ( Idx == nullptr ) {
        std::cout << "CT_CUDA::AllocateData : 'Idx' could not be allocated" << std::endl;
        free(Array);
        free(Idx);
        return false;
    }

    // Randomize the arrays on the host
    srand(time(NULL));
    if ( this->GetBenchType() == CT_PTRCHASE ) { // FIXME: ptrchase looks clunky
        for ( unsigned i = 0; i < idxElems; i++ ) {
            Idx[i] = (uint64_t)(rand()%(idxElems - 1));
        }
    }
    else {
        for ( unsigned i = 0; i < elems; i++ ) {
            Idx[i] = (uint64_t)(rand()%(elems - 1));
        }
    }
    for ( unsigned i=0; i<elems; i++ ) {
        Array[i] = (uint64_t)(rand());
    }

    // allocate data on the target device
    if ( hipMalloc(&d_Array, memSize) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Array' could not be allocated on device" << std::endl;
        hipFree(d_Array);
        free(Array);
        free(Idx);
        return false;
    }

    if ( hipMalloc(&d_Idx, idxMemSize) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Idx' could not be alloced on device" << std::endl;
        hipFree(d_Array);
        hipFree(d_Idx);
        free(Array);
        free(Idx);
        return false;
    }


    // copy arrays from host to target device
    if ( hipMemcpy(d_Array, Array, memSize, hipMemcpyHostToDevice) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Array' could not be copied to device" << std::endl;
        hipFree(d_Array);
        hipFree(d_Idx);
        free(Array);
        free(Idx);
        return false;
    }


    if ( hipMemcpy(d_Idx, Idx, idxMemSize, hipMemcpyHostToDevice) != hipSuccess ) {
        std::cout << "CT_CUDA::AllocateData : 'd_Idx' could not be copied to device" << std::endl;
        hipFree(d_Array);
        hipFree(d_Idx);
        free(Array);
        free(Idx);
        return false;
    }

    return true;
}

bool CT_CUDA::Execute(double &Timing, double &GAMS) {

    CTBaseImpl::CTBenchType BType   = this->GetBenchType(); // benchmark type
    CTBaseImpl::CTAtomType  AType   = this->GetAtomType();  // atomic type
    hipEvent_t start, stop;
    hipEventCreate(start);
    hipEventCreate(stop);
    double OPS       = 0.; // billions of operations

    // determine benchmark type and launch the desired kernel
    if ( BType == CT_RAND ) {
        switch ( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                RAND_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                RAND_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_STRIDE1 ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                STRIDE1_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                STRIDE1_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_STRIDEN ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                STRIDEN_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes, stride );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                STRIDEN_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes, stride );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_PTRCHASE ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                PTRCHASE_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                PTRCHASE_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_SG ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                SG_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(4, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                SG_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(4, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_CENTRAL ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                CENTRAL_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                CENTRAL_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(1, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_SCATTER ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                SCATTER_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(3, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                SCATTER_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(3, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else if ( BType == CT_GATHER ) {
        switch( AType ) {
            case CT_ADD:
                hipEventRecord(start);
                GATHER_ADD<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(3, iters, pes);
                break;
            case CT_CAS:
                hipEventRecord(start);
                GATHER_CAS<<< blocksPerGrid, threadsPerBlock >>>( d_Array, d_Idx, iters, pes );
                hipEventRecord(stop);
                OPS = this->GAM(3, iters, pes);
                break;
            default:
                this->ReportBenchError();
                return false;
                break;
        }
    }
    else {
        this->ReportBenchError();
        return false;
    }
    
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    
    Timing = ms * 1E-3;
    GAMS   = OPS/Timing;

    return true;
}

bool CT_CUDA::FreeData() {
    if ( Array ) {
        free(Array);
    }
    if ( Idx ) {
        free(Idx);
    }
    if ( d_Array ) {
        hipFree(d_Array);
    }
    if ( d_Idx ) {
        hipFree(d_Idx);
    }
    return true;
}

#endif // _CT_CUDA_CUH_

// EOF