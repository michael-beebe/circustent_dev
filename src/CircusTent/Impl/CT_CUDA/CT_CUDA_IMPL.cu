#include "hip/hip_runtime.h"
/*
 * _CT_OMP_IMPL_C_
 *
 * Copyright (C) 2017-2021 Tactical Computing Laboratories, LLC
 * All Rights Reserved
 * contact@tactcomplabs.com
 *
 * See LICENSE in the top level directory for licensing details
 */

#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void RAND_ADD( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes ) {

    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters)
        atomicAdd( &ARRAY[IDX[start]], (u_int64_t) 0x1 );
}

__global__ void RAND_CAS( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes ) {

    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters)
        atomicCAS( &ARRAY[IDX[start]], ARRAY[IDX[start]], ARRAY[IDX[start]]);
}

__global__ void STRIDE1_ADD( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes ) {
    
    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters)
        atomicAdd( &ARRAY[start], (u_int64_t) 0x1 );
}

__global__ void STRIDE1_CAS( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes ) {
    
    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters)
        atomicCAS( &ARRAY[start], ARRAY[start], ARRAY[start]);
}

__global__ void STRIDEN_ADD( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes, uint64_t stride ) {
    
    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters && start % stride == 0 )
        atomicAdd( &ARRAY[start], (u_int64_t) 0x1 );
}

__global__ void STRIDEN_CAS( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes, uint64_t stride ) {
    
    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters && start % stride == 0 )
        atomicCAS( &ARRAY[start], ARRAY[start], ARRAY[start]);
}

__global__ void CENTRAL_ADD( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes ) {
    
    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters)
        atomicAdd( &ARRAY[0], (u_int64_t) 0x1 );
}

__global__ void CENTRAL_CAS( u_int64_t *restrict ARRAY, u_int64_t *restrict IDX, u_int64_t iters, u_int64_t pes ) {
    
    u_int64_t start = threadIdx.x + blockIdx.x * blockDim.x;
    if( start < iters)
        atomicCAS( &ARRAY[0], ARRAY[start], ARRAY[start]);
}